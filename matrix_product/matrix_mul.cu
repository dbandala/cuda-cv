
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <iomanip>
#include <iostream>
#include <random>
#include <stdexcept>
#include <vector>

#define BLOCK_DIM 32

#define checkCuda(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <typename T>
std::vector<T> create_rand_vector(size_t n)
{
    std::random_device r;
    std::default_random_engine e(r());
    std::uniform_int_distribution<int> uniform_dist(-256, 256);

    std::vector<T> vec(n);
    for (size_t i{0}; i < n; ++i)
    {
        vec.at(i) = static_cast<T>(uniform_dist(e));
    }

    return vec;
}

// mat_1: m x n
// mat_2: n x p
// mat_3: m x p
template <typename T>
void mm(T const* mat_1, T const* mat_2, T* mat_3, size_t m, size_t n, size_t p)
{
    // Compute the cells in mat_3 sequentially.
    for (size_t i{0}; i < m; ++i)
    {
        for (size_t j{0}; j < p; ++j)
        {
            T acc_sum{0};
            for (size_t k{0}; k < n; ++k)
            {
                acc_sum += mat_1[i * n + k] * mat_2[k * p + j];
            }
            mat_3[i * p + j] = acc_sum;
        }
    }
}

// mat_1: b x m x n
// mat_2: b x n x p
// mat_3: b x m x p
template <typename T>
void bmm(T const* mat_1, T const* mat_2, T* mat_3, size_t b, size_t m, size_t n,
         size_t p)
{
    // Iterate through the batch dimension.
    for (size_t i{0}; i < b; ++i)
    {
        mm(mat_1 + i * (m * n), mat_2 + i * (n * p), mat_3 + i * (m * p), m, n,
           p);
    }
}

template <typename T>
__global__ void mm_kernel(T const* mat_1, T const* mat_2, T* mat_3, size_t m,
                          size_t n, size_t p)
{
    // 2D block and 2D thread
    // Each thread computes one cell in mat_3.
    size_t i{blockIdx.y * blockDim.y + threadIdx.y};
    size_t j{blockIdx.x * blockDim.x + threadIdx.x};

    // Do not process outside the matrix.
    // Do not forget the equal sign!
    if ((i >= m) || (j >= p))
    {
        return;
    }

    T acc_sum{0};
    for (size_t k{0}; k < n; ++k)
    {
        acc_sum += mat_1[i * n + k] * mat_2[k * p + j];
    }
    mat_3[i * p + j] = acc_sum;
}

// It should be straightforward to extend a kernel to support batching.
template <typename T>
__global__ void bmm_kernel(T const* mat_1, T const* mat_2, T* mat_3, size_t b,
                           size_t m, size_t n, size_t p)
{
    // 2D block and 2D thread
    // Each thread computes one cell in mat_3.
    size_t i{blockIdx.y * blockDim.y + threadIdx.y};
    size_t j{blockIdx.x * blockDim.x + threadIdx.x};

    // Do not process outside the matrix.
    // Do not forget the equal sign!
    if ((i >= m) || (j >= p))
    {
        return;
    }

    // Process the cell of the same index along the batch dimension.
    for (size_t l{0}; l < b; ++l)
    {
        T acc_sum{0};
        for (size_t k{0}; k < n; ++k)
        {
            acc_sum +=
                mat_1[l * m * n + i * n + k] * mat_2[l * n * p + k * p + j];
        }
        mat_3[l * m * p + i * p + j] = acc_sum;
    }
}

template <typename T>
void mm_cuda(T const* mat_1, T const* mat_2, T* mat_3, size_t m, size_t n,
             size_t p)
{
    dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
    dim3 blocks_per_grid(1, 1);
    blocks_per_grid.x = std::ceil(static_cast<double>(p) /
                                  static_cast<double>(threads_per_block.x));
    blocks_per_grid.y = std::ceil(static_cast<double>(m) /
                                  static_cast<double>(threads_per_block.y));
    mm_kernel<<<blocks_per_grid, threads_per_block>>>(mat_1, mat_2, mat_3, m, n,
                                                      p);
}

template <typename T>
void bmm_cuda(T const* mat_1, T const* mat_2, T* mat_3, size_t b, size_t m,
              size_t n, size_t p)
{
    dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
    dim3 blocks_per_grid(1, 1);
    blocks_per_grid.x = std::ceil(static_cast<double>(p) /
                                  static_cast<double>(threads_per_block.x));
    blocks_per_grid.y = std::ceil(static_cast<double>(m) /
                                  static_cast<double>(threads_per_block.y));
    bmm_kernel<<<blocks_per_grid, threads_per_block>>>(mat_1, mat_2, mat_3, b,
                                                       m, n, p);
}

template <typename T>
bool allclose(std::vector<T> const& vec_1, std::vector<T> const& vec_2,
              T const& abs_tol)
{
    if (vec_1.size() != vec_2.size())
    {
        return false;
    }
    for (size_t i{0}; i < vec_1.size(); ++i)
    {
        if (std::abs(vec_1.at(i) - vec_2.at(i)) > abs_tol)
        {
            std::cout << vec_1.at(i) << " " << vec_2.at(i) << std::endl;
            return false;
        }
    }
    return true;
}

template <typename T>
bool random_test_mm_cuda(size_t m, size_t n, size_t p)
{
    std::vector<T> const mat_1_vec{create_rand_vector<T>(m * n)};
    std::vector<T> const mat_2_vec{create_rand_vector<T>(n * p)};
    std::vector<T> mat_3_vec(m * p);
    std::vector<T> mat_4_vec(m * p);
    T const* mat_1{mat_1_vec.data()};
    T const* mat_2{mat_2_vec.data()};
    T* mat_3{mat_3_vec.data()};
    T* mat_4{mat_4_vec.data()};

    mm(mat_1, mat_2, mat_3, m, n, p);

    T *d_mat_1, *d_mat_2, *d_mat_4;

    // Allocate device buffer.
    checkCuda(hipMalloc(&d_mat_1, sizeof(T) * mat_1_vec.size()));
    checkCuda(hipMalloc(&d_mat_2, sizeof(T) * mat_2_vec.size()));
    checkCuda(hipMalloc(&d_mat_4, sizeof(T) * mat_4_vec.size()));

    // Copy data from host to device.
    checkCuda(hipMemcpy(d_mat_1, mat_1, sizeof(T) * mat_1_vec.size(),
                         hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_mat_2, mat_2, sizeof(T) * mat_2_vec.size(),
                         hipMemcpyHostToDevice));

    // Run matrix multiplication on GPU.
    mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p);
    hipDeviceSynchronize();
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Matrix Multiplication kernel failed to execute."
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }

    // Copy data from device to host.
    checkCuda(hipMemcpy(mat_4, d_mat_4, sizeof(T) * mat_4_vec.size(),
                         hipMemcpyDeviceToHost));

    // Free device buffer.
    checkCuda(hipFree(d_mat_1));
    checkCuda(hipFree(d_mat_2));
    checkCuda(hipFree(d_mat_4));

    return allclose<T>(mat_3_vec, mat_4_vec, 1e-4);
}

template <typename T>
bool random_test_bmm_cuda(size_t b, size_t m, size_t n, size_t p)
{
    std::vector<T> const mat_1_vec{create_rand_vector<T>(b * m * n)};
    std::vector<T> const mat_2_vec{create_rand_vector<T>(b * n * p)};
    std::vector<T> mat_3_vec(b * m * p);
    std::vector<T> mat_4_vec(b * m * p);
    T const* mat_1{mat_1_vec.data()};
    T const* mat_2{mat_2_vec.data()};
    T* mat_3{mat_3_vec.data()};
    T* mat_4{mat_4_vec.data()};

    bmm(mat_1, mat_2, mat_3, b, m, n, p);

    T *d_mat_1, *d_mat_2, *d_mat_4;

    // Allocate device buffer.
    checkCuda(hipMalloc(&d_mat_1, sizeof(T) * mat_1_vec.size()));
    checkCuda(hipMalloc(&d_mat_2, sizeof(T) * mat_2_vec.size()));
    checkCuda(hipMalloc(&d_mat_4, sizeof(T) * mat_4_vec.size()));

    // Copy data from host to device.
    checkCuda(hipMemcpy(d_mat_1, mat_1, sizeof(T) * mat_1_vec.size(),
                         hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_mat_2, mat_2, sizeof(T) * mat_2_vec.size(),
                         hipMemcpyHostToDevice));

    // Run matrix multiplication on GPU.
    bmm_cuda(d_mat_1, d_mat_2, d_mat_4, b, m, n, p);
    hipDeviceSynchronize();
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Matrix Multiplication kernel failed to execute."
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }

    // Copy data from device to host.
    checkCuda(hipMemcpy(mat_4, d_mat_4, sizeof(T) * mat_4_vec.size(),
                         hipMemcpyDeviceToHost));

    // Free device buffer.
    checkCuda(hipFree(d_mat_1));
    checkCuda(hipFree(d_mat_2));
    checkCuda(hipFree(d_mat_4));

    return allclose<T>(mat_3_vec, mat_4_vec, 1e-4);
}

template <typename T>
bool random_multiple_test_mm_cuda(size_t num_tests)
{
    std::random_device r;
    std::default_random_engine e(r());
    std::uniform_int_distribution<int> uniform_dist(1, 256);

    size_t m{0}, n{0}, p{0};
    bool success{false};

    for (size_t i{0}; i < num_tests; ++i)
    {
        m = static_cast<size_t>(uniform_dist(e));
        n = static_cast<size_t>(uniform_dist(e));
        p = static_cast<size_t>(uniform_dist(e));
        success = random_test_mm_cuda<T>(m, n, p);
        if (!success)
        {
            return false;
        }
    }

    return true;
}

template <typename T>
bool random_multiple_test_bmm_cuda(size_t num_tests)
{
    std::random_device r;
    std::default_random_engine e(r());
    std::uniform_int_distribution<int> uniform_dist(1, 256);

    size_t b{0}, m{0}, n{0}, p{0};
    bool success{false};

    for (size_t i{0}; i < num_tests; ++i)
    {
        b = static_cast<size_t>(uniform_dist(e));
        m = static_cast<size_t>(uniform_dist(e));
        n = static_cast<size_t>(uniform_dist(e));
        p = static_cast<size_t>(uniform_dist(e));
        success = random_test_bmm_cuda<T>(b, m, n, p);
        if (!success)
        {
            return false;
        }
    }

    return true;
}

template <typename T>
float measure_latency_mm_cuda(size_t m, size_t n, size_t p, size_t num_tests,
                              size_t num_warmups)
{
    hipEvent_t startEvent, stopEvent;
    float time{0.0f};

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    T *d_mat_1, *d_mat_2, *d_mat_4;

    // Allocate device buffer.
    checkCuda(hipMalloc(&d_mat_1, sizeof(T) * m * n));
    checkCuda(hipMalloc(&d_mat_2, sizeof(T) * n * p));
    checkCuda(hipMalloc(&d_mat_4, sizeof(T) * m * p));

    for (size_t i{0}; i < num_warmups; ++i)
    {
        mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p);
    }

    checkCuda(hipEventRecord(startEvent, 0));
    for (size_t i{0}; i < num_tests; ++i)
    {
        mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p);
    }
    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Matrix Multiplication kernel failed to execute."
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
    checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));

    // Free device buffer.
    checkCuda(hipFree(d_mat_1));
    checkCuda(hipFree(d_mat_2));
    checkCuda(hipFree(d_mat_4));

    float latency{time / num_tests};

    return latency;
}

template <typename T>
float measure_latency_bmm_cuda(size_t b, size_t m, size_t n, size_t p,
                               size_t num_tests, size_t num_warmups)
{
    hipEvent_t startEvent, stopEvent;
    float time{0.0f};

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    T *d_mat_1, *d_mat_2, *d_mat_4;

    // Allocate device buffer.
    checkCuda(hipMalloc(&d_mat_1, sizeof(T) * b * m * n));
    checkCuda(hipMalloc(&d_mat_2, sizeof(T) * b * n * p));
    checkCuda(hipMalloc(&d_mat_4, sizeof(T) * b * m * p));

    for (size_t i{0}; i < num_warmups; ++i)
    {
        bmm_cuda(d_mat_1, d_mat_2, d_mat_4, b, m, n, p);
    }

    checkCuda(hipEventRecord(startEvent, 0));
    for (size_t i{0}; i < num_tests; ++i)
    {
        bmm_cuda(d_mat_1, d_mat_2, d_mat_4, b, m, n, p);
    }
    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Matrix Multiplication kernel failed to execute."
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
    checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));

    // Free device buffer.
    checkCuda(hipFree(d_mat_1));
    checkCuda(hipFree(d_mat_2));
    checkCuda(hipFree(d_mat_4));

    float latency{time / num_tests};

    return latency;
}

int main()
{
    constexpr size_t num_tests{10};

    assert(random_multiple_test_mm_cuda<int32_t>(num_tests));
    assert(random_multiple_test_mm_cuda<float>(num_tests));
    assert(random_multiple_test_mm_cuda<double>(num_tests));
    assert(random_multiple_test_bmm_cuda<int32_t>(num_tests));
    assert(random_multiple_test_bmm_cuda<float>(num_tests));
    assert(random_multiple_test_bmm_cuda<double>(num_tests));

    constexpr size_t num_measurement_tests{100};
    constexpr size_t num_measurement_warmups{10};
    //size_t b{128}, m{1024}, n{1024}, p{1024};
    size_t b{64}, m{512}, n{512}, p{512};

    float mm_cuda_int32_latency{measure_latency_mm_cuda<int32_t>(
        m, n, p, num_measurement_tests, num_measurement_warmups)};
    float mm_cuda_float_latency{measure_latency_mm_cuda<float>(
        m, n, p, num_measurement_tests, num_measurement_warmups)};
    float mm_cuda_double_latency{measure_latency_mm_cuda<double>(
        m, n, p, num_measurement_tests, num_measurement_warmups)};

    float bmm_cuda_int32_latency{measure_latency_bmm_cuda<int32_t>(
        b, m, n, p, num_measurement_tests, num_measurement_warmups)};
    float bmm_cuda_float_latency{measure_latency_bmm_cuda<float>(
        b, m, n, p, num_measurement_tests, num_measurement_warmups)};
    float bmm_cuda_double_latency{measure_latency_bmm_cuda<double>(
        b, m, n, p, num_measurement_tests, num_measurement_warmups)};

    std::cout << "Matrix Multiplication CUDA Latency" << std::endl;
    std::cout << "m: " << m << " "
              << "n: " << n << " "
              << "p: " << p << std::endl;
    std::cout << "INT32: " << std::fixed << std::setprecision(5)
              << mm_cuda_int32_latency << " ms" << std::endl;
    std::cout << "FLOAT: " << std::fixed << std::setprecision(5)
              << mm_cuda_float_latency << " ms" << std::endl;
    std::cout << "DOUBLE: " << std::fixed << std::setprecision(5)
              << mm_cuda_double_latency << " ms" << std::endl;

    std::cout << "Batched Matrix Multiplication CUDA Latency" << std::endl;
    std::cout << "b: " << b << " "
              << "m: " << m << " "
              << "n: " << n << " "
              << "p: " << p << std::endl;
    std::cout << "INT32: " << std::fixed << std::setprecision(5)
              << bmm_cuda_int32_latency << " ms" << std::endl;
    std::cout << "FLOAT: " << std::fixed << std::setprecision(5)
              << bmm_cuda_float_latency << " ms" << std::endl;
    std::cout << "DOUBLE: " << std::fixed << std::setprecision(5)
              << bmm_cuda_double_latency << " ms" << std::endl;
}